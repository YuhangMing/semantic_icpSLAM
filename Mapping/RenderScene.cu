#include "hip/hip_runtime.h"
#include "Constant.h"
#include "RenderScene.h"
#include "ParallelScan.h"

#define minMaxSubSample 8
#define renderingBlockSizeX 16
#define renderingBlockSizeY 16

struct Projection {

	int cols, rows;

	Matrix3f RcurrInv;
	float3 tcurr;
	float depthMax, depthMin;
	float fx, fy, cx, cy;

	uint * noRenderingBlocks;
	uint noVisibleBlocks;

	PtrSz<HashEntry> visibleBlocks;
	mutable PtrStepSz<float> zRangeX;
	mutable PtrStep<float> zRangeY;
	mutable PtrSz<RenderingBlock> renderingBlockList;

	__device__ __inline__ float2 project(const float3 & pt3d) const {

		float2 pt2d;
		pt2d.x = fx * pt3d.x / pt3d.z + cx;
		pt2d.y = fy * pt3d.y / pt3d.z + cy;
		return pt2d;
	}

	__device__ __inline__ void atomicMax(float* add, float val) const {
		int* address_as_i = (int*) add;
		int old = *address_as_i, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_i, assumed,
					__float_as_int(fmaxf(val, __int_as_float(assumed))));
		} while (assumed != old);
	}

	__device__ __inline__ void atomicMin(float* add, float val) const {
		int* address_as_i = (int*) add;
		int old = *address_as_i, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_i, assumed,
					__float_as_int(fminf(val, __int_as_float(assumed))));
		} while (assumed != old);
	}

	__device__ __inline__ bool projectBlock(const int3 & pos,
										RenderingBlock & block) const {

		// initialize RenderingBlock
		block.upperLeft = make_short2(zRangeX.cols, zRangeX.rows);
		block.lowerRight = make_short2(-1, -1);
		block.zRange = make_float2(depthMax, depthMin);
		// loop through the 8 corners of a block
		for (int corner = 0; corner < 8; ++corner) {
			// get corner coordinates
			int3 tmp = pos;
			tmp.x += (corner & 1) ? 1 : 0;
			tmp.y += (corner & 2) ? 1 : 0;
			tmp.z += (corner & 4) ? 1 : 0;
			// convert from index to meter
			float3 pt3d = tmp * DeviceMap::BlockSize * DeviceMap::VoxelSize;
			// project the corner from global coordinate system 
			//                 back to camera coordinate system
			pt3d = RcurrInv * (pt3d - tcurr);
			if (pt3d.z < 2e-1)
				continue;
			// project the corner from camera coordinate system
			// 						to image coordinate system
			// and subsample by 8
			float2 pt2d = project(pt3d) / minMaxSubSample;

			if (block.upperLeft.x > floor(pt2d.x))
				block.upperLeft.x = (int) floor(pt2d.x);
			if (block.lowerRight.x < ceil(pt2d.x))
				block.lowerRight.x = (int) ceil(pt2d.x);
			if (block.upperLeft.y > floor(pt2d.y))
				block.upperLeft.y = (int) floor(pt2d.y);
			if (block.lowerRight.y < ceil(pt2d.y))
				block.lowerRight.y = (int) ceil(pt2d.y);
			if (block.zRange.x > pt3d.z)
				block.zRange.x = pt3d.z;
			if (block.zRange.y < pt3d.z)
				block.zRange.y = pt3d.z;
		}
		// check boundary
		if (block.upperLeft.x < 0)
			block.upperLeft.x = 0;
		if (block.upperLeft.y < 0)
			block.upperLeft.y = 0;
		if (block.lowerRight.x >= zRangeX.cols)
			block.lowerRight.x = zRangeX.cols - 1;
		if (block.lowerRight.y >= zRangeX.rows)
			block.lowerRight.y = zRangeX.rows - 1;
		if (block.upperLeft.x > block.lowerRight.x)
			return false;
		if (block.upperLeft.y > block.lowerRight.y)
			return false;
		if (block.zRange.x < depthMin)
			block.zRange.x = depthMin;
		if (block.zRange.y < depthMin)
			return false;

		return true;
	}

	__device__ __inline__ void createRenderingBlockList(int & offset,
			const RenderingBlock & block, int & nx, int & ny) const {

		for (int y = 0; y < ny; ++y)
			for (int x = 0; x < ny; ++x) {
				if (offset < renderingBlockList.size) {
					RenderingBlock & b(renderingBlockList[offset++]);
					b.upperLeft.x = block.upperLeft.x + x * renderingBlockSizeX;
					b.upperLeft.y = block.upperLeft.y + y * renderingBlockSizeY;
					b.lowerRight.x = block.upperLeft.x + renderingBlockSizeX;
					b.lowerRight.y = block.upperLeft.y + renderingBlockSizeY;
					if (b.lowerRight.x > block.lowerRight.x)
						b.lowerRight.x = block.lowerRight.x;
					if (b.lowerRight.y > block.lowerRight.y)
						b.lowerRight.y = block.lowerRight.y;
					b.zRange = block.zRange;
				}
			}
	}

	__device__ __inline__ void operator()() const {

		// get block index in the visibleBlock list
		int x = blockDim.x * blockIdx.x + threadIdx.x;

		bool valid = false;
		uint requiredNoBlocks = 0;
		RenderingBlock block;
		int nx, ny;

		if(x < noVisibleBlocks && visibleBlocks[x].ptr != EntryAvailable) {
			valid = projectBlock(visibleBlocks[x].pos, block);
			// distance in pixels
			float dx = (float) block.lowerRight.x - block.upperLeft.x + 1;
			float dy = (float) block.lowerRight.y - block.upperLeft.y + 1;
			// block size as 16, calculate # of blocks needed
			nx = __float2int_ru(dx / renderingBlockSizeX);
			ny = __float2int_ru(dy / renderingBlockSizeY);
			if (valid) {
				requiredNoBlocks = nx * ny;
				uint totalNoBlocks = *noRenderingBlocks + requiredNoBlocks;
				if (totalNoBlocks >= renderingBlockList.size) {
					requiredNoBlocks = 0;
				}
			}
		}

		int offset = ComputeOffset<1024>(requiredNoBlocks, noRenderingBlocks);
		if (valid && offset != -1 &&
			(offset + requiredNoBlocks) < DeviceMap::MaxRenderingBlocks)
			createRenderingBlockList(offset, block, nx, ny);
	}

	__device__ __inline__ void fillBlocks() const {

		int x = threadIdx.x;
		int y = threadIdx.y;

		int block = blockIdx.x * 4 + blockIdx.y;
		if (block >= renderingBlockList.size)
			return;

		RenderingBlock & b(renderingBlockList[block]);

		int xpos = b.upperLeft.x + x;
		if (xpos > b.lowerRight.x || xpos >= zRangeX.cols)
			return;

		int ypos = b.upperLeft.y + y;
		if (ypos > b.lowerRight.y || ypos >= zRangeX.rows)
			return;

		atomicMin(& zRangeX.ptr(ypos)[xpos], b.zRange.x);
		atomicMax(& zRangeY.ptr(ypos)[xpos], b.zRange.y);

		return;
	}
};

__global__ void projectBlockKernel(const Projection proj) {
	proj();
}

__global__ void fillBlocksKernel(const Projection proj) {
	proj.fillBlocks();
}

__global__ void fillDepthRangeKernel(PtrStepSz<float> range) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x >= range.cols || y >= range.rows)
		return;

	range.ptr(y)[x] = 100;
}

bool CreateRenderingBlocks(const DeviceArray<HashEntry> & visibleBlocks,
						  DeviceArray2D<float> & zRangeX,
						  DeviceArray2D<float> & zRangeY,
						  const float & depthMax,
						  const float & depthMin,
						  DeviceArray<RenderingBlock> & renderingBlockList,
						  DeviceArray<uint> & noRenderingBlocks,
						  Matrix3f RviewInv,
						  float3 tview,
						  uint noVisibleBlocks,
						  float fx,
						  float fy,
						  float cx,
						  float cy) {

	if(noVisibleBlocks == 0)
		return false;

	int cols = zRangeX.cols;
	int rows = zRangeX.rows;
	noRenderingBlocks.clear();

	Projection proj;
	proj.fx = fx;
	proj.fy = fy;
	proj.cx = cx;
	proj.cy = cy;
	proj.visibleBlocks = visibleBlocks;
	proj.cols = cols;
	proj.rows = rows;
	proj.RcurrInv = RviewInv;
	proj.tcurr = tview;
	proj.zRangeX = zRangeX;
	proj.zRangeY = zRangeY;
	proj.depthMax = depthMax;
	proj.depthMin = depthMin;
	proj.noRenderingBlocks = noRenderingBlocks;
	proj.noVisibleBlocks = noVisibleBlocks;
	proj.renderingBlockList = renderingBlockList;

	dim3 block, thread;
	thread = dim3(16, 4);
	block.x = DivUp(cols, thread.x);
	block.y = DivUp(rows, thread.y);

	zRangeY.clear();
	// subsampled by 3 times.480x640 / 2/2/2 = 60x80
	float zRangeMax[60][80];
	for(int i = 0; i < 80; ++i) {
		for(int j = 0; j < 60; ++j) {
			zRangeMax[j][i] = 100.f;
		}
	}
	zRangeX.upload(zRangeMax);

	thread = dim3(1024);
	block = dim3(DivUp((int) noVisibleBlocks, block.x));

	projectBlockKernel<<<block, thread>>>(proj);

	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	uint totalBlocks;
	noRenderingBlocks.download((void*) &totalBlocks);

	if (totalBlocks == 0) {
		return false;
	}

	thread = dim3(16, 16);
	block = dim3((uint)ceil((float)totalBlocks / 4), 4);

	// store the zRange min and max of a block into corresponding place of array
	fillBlocksKernel<<<block, thread>>>(proj);
	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());

	return true;
}

struct Rendering {

	int cols, rows;
	DeviceMap map;
	mutable PtrStep<float4> vmap;
	mutable PtrStep<float4> nmap;
	PtrStep<float> zRangeX;
	PtrStep<float> zRangeY;
	float invfx, invfy, cx, cy;
	Matrix3f Rview, RviewInv;
	float3 tview;

	__device__ __inline__ float readSdf(const float3 & pt3d, HashEntry & cache, bool & valid) {
		Voxel voxel = map.FindVoxel(pt3d, cache, valid);
		if (voxel.weight == 0)
			valid = false;
		return voxel.sdf;
	}

	__device__ __inline__ float readSdfInterped(const float3 & pt, HashEntry & cache, bool & valid) {
		// Trilinear Interpolation

		float3 xyz = pt - floor(pt);
		// sdf in current voxel grid and next voxel grid
		float sdf[2], result[4];

		// linear interpolation along x-axis, y=0, z=0
		sdf[0] = map.FindVoxel(pt, cache, valid).sdf;
		sdf[1] = map.FindVoxel(pt + make_float3(1, 0, 0), cache, valid).sdf;
		result[0] = (1.0f - xyz.x) * sdf[0] + xyz.x * sdf[1];

		// linear interpolation along x-axis, y=1, z=0
		sdf[0] = map.FindVoxel(pt + make_float3(0, 1, 0), cache, valid).sdf;
		sdf[1] = map.FindVoxel(pt + make_float3(1, 1, 0), cache, valid).sdf;
		result[1] = (1.0f - xyz.x) * sdf[0] + xyz.x * sdf[1];
		// bilinear interpolation along xy-plane, z=0
		result[2] = (1.0f - xyz.y) * result[0] + xyz.y * result[1];

		// linear interpolation along x-axis, y=0, z=1
		sdf[0] = map.FindVoxel(pt + make_float3(0, 0, 1), cache, valid).sdf;
		sdf[1] = map.FindVoxel(pt + make_float3(1, 0, 1), cache, valid).sdf;
		result[0] = (1.0f - xyz.x) * sdf[0] + xyz.x * sdf[1];

		// linear interpolation along x-axis, y=1, z=1
		sdf[0] = map.FindVoxel(pt + make_float3(0, 1, 1), cache, valid).sdf;
		sdf[1] = map.FindVoxel(pt + make_float3(1, 1, 1), cache, valid).sdf;
		result[1] = (1.0f - xyz.x) * sdf[0] + xyz.x * sdf[1];
		// bilinear interpolation along xy-plane, z=1
		result[3] = (1.0f - xyz.y) * result[0] + xyz.y * result[1];

		// return trilinear interpolation result
		return (1.0f - xyz.z) * result[2] + xyz.z * result[3];
	}

	__device__ __inline__ bool readNormal(const float3 & pt, HashEntry & cache, float3 & n) {
		// gradients on 3d space

		bool valid;
		float sdf[6];
		sdf[0] = readSdfInterped(pt + make_float3(1, 0, 0), cache, valid);
		if(isnan(sdf[0]) || sdf[0] == 1.0f || !valid)
			return false;

		sdf[1] = readSdfInterped(pt + make_float3(-1, 0, 0), cache, valid);
		if(isnan(sdf[1]) || sdf[1] == 1.0f || !valid)
			return false;

		sdf[2] = readSdfInterped(pt + make_float3(0, 1, 0), cache, valid);
		if(isnan(sdf[2]) || sdf[2] == 1.0f || !valid)
			return false;

		sdf[3] = readSdfInterped(pt + make_float3(0, -1, 0), cache, valid);
		if(isnan(sdf[3]) || sdf[3] == 1.0f || !valid)
			return false;

		sdf[4] = readSdfInterped(pt + make_float3(0, 0, 1), cache, valid);
		if(isnan(sdf[4]) || sdf[4] == 1.0f || !valid)
			return false;

		sdf[5] = readSdfInterped(pt + make_float3(0, 0, -1), cache, valid);
		if(isnan(sdf[5]) || sdf[5] == 1.0f || !valid)
			return false;

		// differentiate
		n = make_float3(sdf[0] - sdf[1], sdf[2] - sdf[3], sdf[4] - sdf[5]);
		n = normalised(RviewInv * n);
		return true;
	}

	__device__ __inline__ void operator()() {

		// get pixel coordinate
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int y = blockDim.y * blockIdx.y + threadIdx.y;
		if (x >= cols || y >= rows)
			return;

		// initialize vmap and nmap
		vmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));
		nmap.ptr(y)[x] = make_float4(__int_as_float(0x7fffffff));

		// subsample by 8
		int2 locId;
		locId.x = __float2int_rd((float) x / minMaxSubSample);
		locId.y = __float2int_rd((float) y / minMaxSubSample);

		// get range in the z direction (minDepth, maxDepth)
		// every 8 pixels share a z range
		float2 zRange;
		zRange.x = zRangeX.ptr(locId.y)[locId.x];
		zRange.y = zRangeY.ptr(locId.y)[locId.x];
		if(zRange.y < 1e-3 || zRange.x < 1e-3 || isnan(zRange.x) || isnan(zRange.y))
			return;

		float sdf = 1.0f;
		// calculate the ray direction and start point, in voxel grid
		float3 pt3d;
		pt3d.z = zRange.x;
		pt3d.x = pt3d.z * ((float) x - cx) * invfx;
		pt3d.y = pt3d.z * ((float) y - cy) * invfy;
		float dist_s = norm(pt3d) * DeviceMap::voxelSizeInv;
		float3 block_s = (Rview * pt3d + tview) * DeviceMap::voxelSizeInv;

		pt3d.z = zRange.y;
		pt3d.x = pt3d.z * ((float) x - cx) * invfx;
		pt3d.y = pt3d.z * ((float) y - cy) * invfy;
		float dist_e = norm(pt3d) * DeviceMap::voxelSizeInv;
		float3 block_e = (Rview * pt3d + tview) * DeviceMap::voxelSizeInv;

		float3 dir = normalised(block_e - block_s);
		float3 result = block_s;

		bool valid_sdf = false;
		bool found_pt = false;
		float step;
		HashEntry b;
		float prev_sdf;
		// loop until zero-corssing is found
		while (dist_s < dist_e) {
			prev_sdf = sdf;
			sdf = readSdf(result, b, valid_sdf);
			if(!valid_sdf) {
				step = DeviceMap::BlockSize;
			}
			else {
				// find a small range aroung 0, to interpolate more precise sdf
				if (sdf <= 0.1f && sdf >= -0.5f) {
					sdf = readSdfInterped(result, b, valid_sdf);
				}

				// break if: 1) start from negative area -> invalid surface measurement
				// 			 2) zero-corssing found
				if (prev_sdf < 0.f && sdf >= 0.f) //Zero crossing from behind
                    return;
				if (prev_sdf > 0.f && sdf <= 0.f) //Zero crossing from front
					break;

				// if (!isnan(sdf)){
				// 	// why sdf*stepScale: Kind of adaptive search
				// 	// larger sdf means far away from zero-crossing, larger step would be preferred
				// 	// vice versa
				// 	step = max(sdf * DeviceMap::stepScale, 1.0f);
				// }
				// else{
				// 	step = DeviceMap::BlockSize;
				// }
				step = 1.0;
			}

			result += step * dir;
			dist_s += step;
		}

		// find specific zero-crossing ponit
		if(sdf <= 0.0f) {
			step = sdf * DeviceMap::stepScale;
			result += step * dir;

			sdf = readSdfInterped(result, b, valid_sdf);

			step = sdf * DeviceMap::stepScale;
			result += step * dir;
			found_pt = true;
		}

		// calculate normal and vertex map if zero-crossing point is found
		if(found_pt) {
			float3 normal;
			if(readNormal(result, b, normal)) {

				// project from voxel grid to meters in world coordinate system
				result = RviewInv * (result * DeviceMap::VoxelSize - tview);

				vmap.ptr(y)[x] = make_float4(result, 1.0);
				nmap.ptr(y)[x] = make_float4(normal, 1.0);
			}
		}
	}
};

__global__ void __launch_bounds__(32, 16) RayCastKernel(Rendering cast) {
	cast();
}

void Raycast(DeviceMap map,
			 DeviceArray2D<float4> & vmap,
			 DeviceArray2D<float4> & nmap,
			 DeviceArray2D<float> & zRangeX,
			 DeviceArray2D<float> & zRangeY,
			 Matrix3f Rview,
			 Matrix3f RviewInv,
			 float3 tview,
			 float invfx,
			 float invfy,
			 float cx,
			 float cy) {

	int cols = vmap.cols;
	int rows = vmap.rows;

	Rendering cast;
	cast.cols = cols;
	cast.rows = rows;

	cast.map = map;
	cast.vmap = vmap;
	cast.nmap = nmap;
	cast.zRangeX = zRangeX;
	cast.zRangeY = zRangeY;
	cast.invfx = invfx;
	cast.invfy = invfy;
	cast.cx = cx;
	cast.cy = cy;
	cast.Rview = Rview;
	cast.RviewInv = RviewInv;
	cast.tview = tview;

	dim3 block;
	dim3 thread(4, 8);
	block.x = DivUp(cols, thread.x);
	block.y = DivUp(rows, thread.y);

	RayCastKernel<<<block, thread>>>(cast);

	SafeCall(hipGetLastError());
	SafeCall(hipDeviceSynchronize());
}
